#include "hip/hip_runtime.h"
// @file vl_nnpool.cu
// @brief Pooling block MEX wrapper
// @author Andrea Vedaldi
// @author Karel Lenc

/*
Copyright (C) 2014-15 Andrea Vedaldi and Karel Lenc.
All rights reserved.

This file is part of the VLFeat library and is made available under
the terms of the BSD license (see the COPYING file).
*/

#include "bits/mexutils.h"
#include "bits/datamex.hpp"
#include "bits/nncircpooling_mask.hpp"

#if ENABLE_GPU
#include "bits/datacu.hpp"
#endif

#include <assert.h>

/* option codes */
enum {
  opt_overlap = 0,
  opt_pad,
  opt_method,
  opt_verbose,
  opt_cudnn,
  opt_no_cudnn,
} ;

/* options */
VLMXOption  options [] = {
  {"Overlap",           1,   opt_overlap            },
  {"Pad",              1,   opt_pad               },
  {"Method",           1,   opt_method            },
  {"Verbose",          1,   opt_verbose           },
  {"CUDNN",            0,   opt_cudnn             },
  {"NoCUDNN",          0,   opt_no_cudnn          },
  {0,                  0,   0                     }
} ;

/* ---------------------------------------------------------------- */
/*                                                          Context */
/* ---------------------------------------------------------------- */

vl::MexContext context ;

/*
 Resetting the context here resolves a crash when MATLAB quits and
 the ~Context function is implicitly called on unloading the MEX file.
 */
void atExit()
{
  context.clear() ;
}

/* ---------------------------------------------------------------- */
/*                                                       MEX driver */
/* ---------------------------------------------------------------- */

enum {
  IN_DATA = 0, IN_COORDS, IN_SIZE, IN_DEROUTPUT, IN_END
} ;

enum {
  OUT_RESULT = 0, OUT_END
} ;

void mexFunction(int nout, mxArray *out[],
                 int nin, mxArray const *in[])
{
  int poolAngles ;
  int poolRings ;
  float overlapRing = 0 ;
  float overlapAngle = 0 ;
  int padLeft = 0 ;
  int padRight = 0 ;
  int padTop = 0 ;
  int padBottom = 0 ;
  vl::PoolingMethod method = vl::vlCircPoolingMaskMax ;
  bool backMode = false ;

  int verbosity = 0 ;
  int opt ;
  int next = IN_END ;
  mxArray const *optarg ;

  /* -------------------------------------------------------------- */
  /*                                            Check the arguments */
  /* -------------------------------------------------------------- */

  mexAtExit(atExit) ;

  if (nin < 3) {
    mexErrMsgTxt("The arguments are less than three.") ;
  }

  if (nin > 3 && vlmxIsString(in[3],-1)) {
    next = 3 ;
    backMode = 0 ;
  } else {
    backMode = (nin >= 4) ;
  }

  while ((opt = vlmxNextOption (in, nin, options, &next, &optarg)) >= 0) {


    switch (opt) {
      case opt_verbose :
        ++ verbosity ;
        break ;

      case opt_overlap :
        if (!vlmxIsPlainMatrix(optarg,-1,-1)) {
          mexErrMsgTxt("STRIDE is not a plain matrix.") ;
        }
        switch (mxGetNumberOfElements(optarg)) {
          case 1:
        	  overlapRing = (float)mxGetPr(optarg)[0] ;
        	  overlapAngle = overlapRing ;
            break ;
          case 2:
        	  overlapRing = (float)mxGetPr(optarg)[0] ;
        	  overlapAngle = (float)mxGetPr(optarg)[1] ;
            break ;
          default:
            mexErrMsgTxt("OVERLAP has neither one nor two elements.") ;
        }
        break ;

      case opt_pad :
        if (!vlmxIsPlainMatrix(optarg,-1,-1)) {
          mexErrMsgTxt("PAD is not a plain matrix.") ;
        }
        switch (mxGetNumberOfElements(optarg)) {
          case 1:
            padLeft = (int)mxGetPr(optarg)[0] ;
            padRight = padLeft ;
            padTop = padLeft ;
            padBottom = padLeft ;
            break ;
          case 4:
            padTop = (int)mxGetPr(optarg)[0] ;
            padBottom = (int)mxGetPr(optarg)[1] ;
            padLeft = (int)mxGetPr(optarg)[2] ;
            padRight = (int)mxGetPr(optarg)[3] ;
            break ;
          default:
            mexErrMsgTxt("PAD has neither one nor four elements.") ;
        }
        break;

      case opt_method :
        if (!vlmxIsString(optarg,-1)) {
           vlmxError(VLMXE_IllegalArgument, "METHOD is not a string.") ;
        }
        if (vlmxIsEqualToStringI(optarg, "max")) {
          method = vl::vlCircPoolingMaskMax ;
        } else if (vlmxIsEqualToStringI(optarg, "avg")) {
          method = vl::vlCircPoolingMaskAverage ;
        } else {
          vlmxError(VLMXE_IllegalArgument, "METHOD is not a supported method.") ;
        }
        break;

      case opt_no_cudnn :
#if ENABLE_CUDNN
        context.getCudaHelper().setCudnnEnabled(false) ;
#endif
        break ;

      case opt_cudnn :
#if ENABLE_CUDNN
        context.getCudaHelper().setCudnnEnabled(true) ;
#endif
        break ;

      default:
        break ;
    }
  }

  vl::MexTensor data(context) ;
  vl::MexTensor pcoords(context) ;
  vl::MexTensor derOutput(context) ;

  data.init(in[IN_DATA]) ;
  data.reshape(4) ; // -> 4 dimensions
  pcoords.init(in[IN_COORDS]);
  pcoords.reshape(4);
  if (backMode) {
    derOutput.init(in[IN_DEROUTPUT]) ;
    derOutput.reshape(4) ; // -> 4 dimensions
  }

  if (backMode && ! vl::areCompatible(data, derOutput)) {
    mexErrMsgTxt("DATA and DEROUTPUT do not have compatible formats.") ;
  }

  if (!vlmxIsPlainMatrix(in[IN_SIZE],-1,-1)) {
    mexErrMsgTxt("SIZE is not a plain matrix.") ;
  }
  switch (mxGetNumberOfElements(in[IN_SIZE])) {
    case 1:
      poolRings = mxGetPr(in[IN_SIZE])[0] ;
      poolAngles = 1;
      break ;
    case 2:
      poolRings = mxGetPr(in[IN_SIZE])[0] ;
      poolAngles = mxGetPr(in[IN_SIZE])[1] ;
      break ;
    default:
      mexErrMsgTxt("SIZE has neither one nor two elements.") ;
  }

  if (poolRings == 0 || poolAngles == 0) {
    mexErrMsgTxt("A dimension of the pooling SIZE is void.") ;
  }
 /* if (data.getHeight() + (padTop+padBottom) < poolHeight ||
      data.getWidth() + (padLeft+padRight) < poolWidth) {
    mexErrMsgTxt("The pooling window is larger than the DATA (including padding).") ;
  }
  if (padLeft < 0 ||
      padRight < 0 ||
      padTop < 0 ||
      padBottom < 0) {
    mexErrMsgTxt("An element of PAD is negative.") ;
  }*/
  /*if (padLeft >= poolWidth ||
      padRight >= poolWidth ||
      padTop >= poolHeight  ||
      padBottom >= poolHeight) {
    mexErrMsgTxt("A padding value is larger or equal than the size of the pooling window.") ;
  }*/

  /* Get the output Shape */
  vl::TensorShape outputShape(poolRings,poolAngles,data.getDepth(),data.getSize());

  if (backMode && (derOutput != outputShape)) {
    mexErrMsgTxt("DEROUTPUT dimensions are incompatible with X and POOL.") ;
  }

  /* Create output buffers */
  vl::DeviceType deviceType = data.getDeviceType() ;
  vl::DataType dataType = data.getDataType() ;
  vl::MexTensor output(context) ;
  vl::MexTensor derData(context) ;

  if (!backMode) {
    output.initWithZeros(deviceType, dataType, outputShape) ;
  } else {
    derData.initWithZeros(deviceType, dataType, data.getShape()) ;
  }

  if (verbosity > 0) {
    mexPrintf("vl_nncircpoolmask: %s; %s", backMode?"backward":"forward", (data.getDeviceType()==vl::VLDT_GPU) ? "VLDT_GPU" : "CPU") ;
    if (data.getDeviceType() == vl::VLDT_GPU) {
#if ENABLE_CUDNN
      mexPrintf("; %s\n", context.getCudaHelper().getCudnnEnabled() ? "cuDNN" : "MatConvNet") ;
#else
      mexPrintf("; MatConvNet\n") ;
#endif
    } else {
      mexPrintf("; MatConvNet\n") ;
    }
    mexPrintf("vl_nncircpoolmask: overlap: [%d %d], pad: [%d %d %d %d]\n",
              overlapRing, overlapAngle,
              padTop, padBottom, padLeft, padRight) ;
    vl::print("vl_nncircpoolmask: data: ", data) ;
    mexPrintf("vl_nncircpoolmask: pooling: %d x %d\n", poolRings, poolAngles);
    mexPrintf("vl_nncircpoolmask: method: %s\n", (method == vl::vlCircPoolingMaskMax) ? "max" : "avg") ;
    if (backMode) {
      vl::print("vl_nncircpoolmask:: derOutput: ", derOutput) ;
      vl::print("vl_nncircpoolmask:: derData: ", derData) ;
    } else {
      vl::print("vl_nncircpoolmask:: output: ", output) ;
    }
  }

  /* -------------------------------------------------------------- */
  /*                                                    Do the work */
  /* -------------------------------------------------------------- */

  vl::ErrorCode error ;
  if (!backMode) {
	//mexPrintf("forward vl_nncircpoolmask: pooling: %d x %d\n", poolRings, poolAngles);
    error = vl::nncircpooling_mask_forward(context,
                                  output, data,
                                  pcoords,
                                  method,
                                  poolRings,poolAngles,
								  overlapRing, overlapAngle,
                                  padTop, padBottom, padLeft, padRight) ;
  } else {
	//mexPrintf("backward vl_nncircpoolmask: pooling: %d x %d\n", poolRings, poolAngles);
    error = vl::nncircpooling_mask_backward(context,
                                   derData, data, pcoords,
                                   derOutput,
                                   method,
                                   poolRings,poolAngles,
								   overlapRing, overlapAngle,
                                   padTop, padBottom, padLeft, padRight) ;
  }

  /* -------------------------------------------------------------- */
  /*                                                         Finish */
  /* -------------------------------------------------------------- */

  if (error != vl::VLE_Success) {
    mexErrMsgTxt(context.getLastErrorMessage().c_str()) ;
  }
  if (backMode) {
    out[OUT_RESULT] = derData.relinquish() ;
  } else {
    out[OUT_RESULT] = output.relinquish() ;
  }
}
